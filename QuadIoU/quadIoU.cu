#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

struct Point {
    double x, y;
};

struct Edge {
    struct Point startPoint;
    struct Point endPoint;
};

struct Quadrilateral {
    Point corners[4];
};

__device__ int findMaxQuadCoordinate(Quadrilateral& box, char x_or_y){
    // Find the maximum x-coordinate of the quadrilateral
    if (x_or_y == 'x'){
        int max_x = box.corners[0].x;
        for (int i = 1; i < 4; ++i) {
            if (box.corners[i].x > max_x) {
                max_x = box.corners[i].x;
            }
        }
        return max_x;
    } else if (x_or_y == 'y'){
        int max_y = box.corners[0].y;
        for (int i = 1; i < 4; ++i) {
            if (box.corners[i].y > max_y) {
                max_y = box.corners[i].y;
            }
        }
        return max_y;
    } else {
        return -1;
    }
}

__device__ int orientation(Point p, Point q, Point r) {
    double val = (q.y - p.y) * (r.x - q.x) - (q.x - p.x) * (r.y - q.y);

    if (abs(val) < 1e-10) return 0;  // colinear
    return (val > 0) ? 1 : 2;  // clock or counterclock wise
}

__device__ int isPointInsideQuadrilateral(Point point_to_check, Quadrilateral box) {
    // Find the maximum x-coordinate of the quadrilateral
    int max_x = findMaxQuadCoordinate(box, 'x');
    int max_y = findMaxQuadCoordinate(box, 'y');
    // If the point's x-coordinate is greater than the max x-coordinate, it's outside
    if (point_to_check.x > max_x) return -1;
    if (point_to_check.y > max_y) return -1;

    // For each edge of the quadrilateral
    for (int i = 0; i < 4; i++) {
        // Get the current edge's start and end points
        Point start_point = box.corners[i];
        Point end_point = box.corners[(i + 1) % 4]; // Wrap around to the first point after the last
        // Calculate the cross product to determine where the point is in relation to the edge
        double cross_product = (start_point.y - point_to_check.y) * (end_point.x - point_to_check.x) -
                                (start_point.x - point_to_check.x) * (end_point.y - point_to_check.y);
        if (cross_product > 0) {
            return -1; // Point is outside the quadrilateral
        }
        else if (cross_product == 0) {
            return 0; // Point is on the boundary of the quadrilateral
        }
    }
    return 1; // Point is inside the quadrilateral
}

__device__ bool onSegment(Point p, Point q, Point r) {
    return q.x <= max(p.x, r.x) && q.x >= min(p.x, r.x) &&
           q.y <= max(p.y, r.y) && q.y >= min(p.y, r.y);
}

// Helper function to check if two given points are equal
__device__ bool arePointsEqual(const Point& p1, const Point& p2) {
    return abs(p1.x - p2.x) < 1e-10 && abs(p1.y - p2.y) < 1e-10;
}

__device__ bool doIntersect(Point p1, Point q1, Point p2, Point q2, Point& intersection) {
    // Find the four orientations needed for general and
    // special cases
    int o1 = orientation(p1, q1, p2);
    int o2 = orientation(p1, q1, q2);
    int o3 = orientation(p2, q2, p1);
    int o4 = orientation(p2, q2, q1);

    // General case
    if (o1 != o2 && o3 != o4) {
        // Line AB represented as a1x + b1y = c1
        double a1 = q1.y - p1.y;
        double b1 = p1.x - q1.x;
        double c1 = a1 * (p1.x) + b1 * (p1.y);

        // Line CD represented as a2x + b2y = c2
        double a2 = q2.y - p2.y;
        double b2 = p2.x - q2.x;
        double c2 = a2 * (p2.x) + b2 * (p2.y);

        double determinant = a1 * b2 - a2 * b1;

        if (abs(determinant) < 1e-10) {
            return false; // The lines are parallel. This is simplified
                          // by returning false here.
        } else {
            intersection.x = (b2 * c1 - b1 * c2) / determinant;
            intersection.y = (a1 * c2 - a2 * c1) / determinant;
            return true;
        }
    }

    // Special Cases
    // p1, q1 and p2 are colinear and p2 lies on segment p1q1
    if (o1 == 0 && onSegment(p1, p2, q1)) {
        intersection = p2;
        return true;
    }

    // p1, q1 and q2 are colinear and q2 lies on segment p1q1
    if (o2 == 0 && onSegment(p1, q2, q1)) {
        intersection = q2;
        return true;
    }

    // p2, q2 and p1 are colinear and p1 lies on segment p2q2
    if (o3 == 0 && onSegment(p2, p1, q2)) {
        intersection = p1;
        return true;
    }

    // p2, q2 and q1 are colinear and q1 lies on segment p2q2
    if (o4 == 0 && onSegment(p2, q1, q2)) {
        intersection = q1;
        return true;
    }

    return false; // Doesn't fall in any of the above cases
}

__device__ int findIntersectionPoints(Quadrilateral quad_1, Quadrilateral quad_2, Point* intersections, int maxIntersections) {
    int numIntersections = 0;
    for (int i = 0; i < 4; ++i) {
        Edge e1 = {quad_1.corners[i], quad_1.corners[(i + 1) % 4]};
        for (int j = 0; j < 4; ++j) {
            Edge e2 = {quad_2.corners[j], quad_2.corners[(j + 1) % 4]};
            Point intersection;
            if (doIntersect(e1.startPoint, e1.endPoint, e2.startPoint, e2.endPoint, intersection)) {
                // Check if this intersection is already in the intersections array
                bool alreadyExists = false;
                for (int k = 0; k < numIntersections; ++k) {
                    if (arePointsEqual(intersections[k], intersection)) {
                        alreadyExists = true;
                        break;
                    }
                }
                if (!alreadyExists && numIntersections < maxIntersections) {
                    intersections[numIntersections++] = intersection;
                }
            }
        }
    }
    return numIntersections;
}

__device__ int findPointInside(Quadrilateral quad_1, Quadrilateral quad_2, Point* inside_points, int maxPoints) {
    int numInsidePoints = 0;

    for (int i = 0; i < 4; i++) {
        if (isPointInsideQuadrilateral(quad_1.corners[i], quad_2) == 1) {
            if (numInsidePoints < maxPoints) {
                inside_points[numInsidePoints++] = quad_1.corners[i];
            }
        }
        if (isPointInsideQuadrilateral(quad_2.corners[i], quad_1) == 1) {
            if (numInsidePoints < maxPoints) {
                inside_points[numInsidePoints++] = quad_2.corners[i];
            }
        }
    }

    return numInsidePoints;
}

__device__ Point findCentroid(const Point* points, int numPoints) {
    Point centroid = {0, 0};
    if (numPoints <= 0) {
        return centroid; // Return the origin if there are no points
    }

    for (int i = 0; i < numPoints; ++i) {
        centroid.x += points[i].x;
        centroid.y += points[i].y;
    }
    centroid.x /= numPoints;
    centroid.y /= numPoints;
    return centroid;
}

__device__ double computeAngle(const Point& centroid, const Point& p) {
    return atan2(p.y - centroid.y, p.x - centroid.x);
}

__device__ bool comparePoints(const Point& p1, const Point& p2, const Point& centroid) {
    double angle1 = computeAngle(centroid, p1);
    double angle2 = computeAngle(centroid, p2);

    // If angles are the same, sort based on distance from centroid
    if (angle1 == angle2) {
        double dist1 = (p1.x - centroid.x) * (p1.x - centroid.x) +
                       (p1.y - centroid.y) * (p1.y - centroid.y);
        double dist2 = (p2.x - centroid.x) * (p2.x - centroid.x) +
                       (p2.y - centroid.y) * (p2.y - centroid.y);
        return dist1 < dist2;
    }
    return angle1 < angle2;
}

__device__ void sortPointsClockwise(Point* points, int numPoints) {
    // First, find the centroid
    Point centroid = findCentroid(points, numPoints);

    // Perform selection sort on the array of points
    for (int i = 0; i < numPoints - 1; i++) {
        // Find the minimum element in unsorted array
        int minIdx = i;
        for (int j = i + 1; j < numPoints; j++) {
            if (comparePoints(points[j], points[minIdx], centroid)) {
                minIdx = j;
            }
        }
        // Swap the found minimum element with the first element
        Point temp = points[minIdx];
        points[minIdx] = points[i];
        points[i] = temp;
    }
}

__device__ void sortPointsClockwise(Quadrilateral& quad) {
    // Directly use the corners array of the quadrilateral for sorting
    sortPointsClockwise(quad.corners, 4);
}

__device__ double polygonArea(const Point* polygon, int n) {
    double area = 0.0;
    // Calculate the sum for the Shoelace formula
    for (int i = 0; i < n - 1; ++i) {
        area += (polygon[i].x * polygon[i + 1].y - polygon[i + 1].x * polygon[i].y);
    }
    // Add the last term (closing the polygon loop)
    area += (polygon[n - 1].x * polygon[0].y - polygon[0].x * polygon[n - 1].y);
    return fabs(area) / 2.0;
}

__device__ double polygonArea(const Quadrilateral& quad) {
    // Since a quadrilateral always has 4 points, we pass the corners directly
    return polygonArea(quad.corners, 4);
}

__device__ double intersectionArea(Quadrilateral quad0, Quadrilateral quad1) {
    const int MAX_INTERSECTIONS = 8; // 8 intersections max
    const int MAX_TOTAL_POINTS = MAX_INTERSECTIONS * 2; // Intersection + Inside points
    Point intersectionPoints[MAX_INTERSECTIONS];
    Point insidePoints[MAX_INTERSECTIONS];
    Point allPoints[MAX_TOTAL_POINTS];
    int numIntersectionPoints = 0;
    int numInsidePoints = 0;
    int totalPoints = 0;

    // Sort the corners of both quadrilaterals
    sortPointsClockwise(quad0);
    sortPointsClockwise(quad1);

    // Find intersection points
    numIntersectionPoints = findIntersectionPoints(quad0, quad1, intersectionPoints, MAX_INTERSECTIONS);

    // Find points inside the other quadrilateral
    numInsidePoints = findPointInside(quad0, quad1, insidePoints, MAX_INTERSECTIONS);

    // Combine intersection points with inside points
    for (int i = 0; i < numIntersectionPoints; ++i) {
        allPoints[totalPoints++] = intersectionPoints[i];
    }
    for (int i = 0; i < numInsidePoints; ++i) {
        allPoints[totalPoints++] = insidePoints[i];
    }

    // Sort all points to prepare for area calculation
    sortPointsClockwise(allPoints, totalPoints);

    // Calculate the area of the polygon formed by the points
    double intersectArea = polygonArea(allPoints, totalPoints);

    return intersectArea;
}

__device__ double unionArea(Quadrilateral quad0, Quadrilateral quad1, double intersect_area){
    return polygonArea(quad0) + polygonArea(quad1) - intersect_area;
}

__device__ double calculateIoU(double intersect_area, double union_area){
    return intersect_area / union_area;
}

__global__ void computeIoUKernel(Quadrilateral *quads1, Quadrilateral *quads2, double *output, int n1, int n2) {
    int idx1 = blockIdx.x * blockDim.x + threadIdx.x;
    int idx2 = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx1 < n1 && idx2 < n2) {
        int outputIndex = idx1 * n2 + idx2;
        double areaIntersection = intersectionArea(quads1[idx1], quads2[idx2]);
        double areaUnion = unionArea(quads1[idx1], quads2[idx2], areaIntersection);
        output[outputIndex] = calculateIoU(areaIntersection, areaUnion);
    }
}

const int NUM_QUAD0 = 200;
const int NUM_QUAD1 = 300;

int main() {
    hipError_t err;
    // Allocate memory on the host using Unified Memory (accessible by both CPU and GPU)
    Quadrilateral *quads0, *quads1;
    err = hipMallocManaged(&quads0, NUM_QUAD0 * sizeof(Quadrilateral));
    if (err != hipSuccess) {
        std::cerr << "CUDA error (hipMallocManaged for quads0): " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    err = hipMallocManaged(&quads1, NUM_QUAD1 * sizeof(Quadrilateral));
    if (err != hipSuccess) {
        std::cerr << "CUDA error (hipMallocManaged for quads1): " << hipGetErrorString(err) << std::endl;
        hipFree(quads0); // Clean up previously allocated memory
        return -1;
    }
    hipMallocManaged(&quads0, NUM_QUAD0 * sizeof(Quadrilateral));
    hipMallocManaged(&quads1, NUM_QUAD1 * sizeof(Quadrilateral));

    // Initialize the arrays with test quadrilaterals
    for (int i = 0; i < NUM_QUAD0; ++i) {
        quads0[i].corners[0] = {0.0, 0.0};
        quads0[i].corners[1] = {1.0, 0.0};
        quads0[i].corners[2] = {1.0, 1.0};
        quads0[i].corners[3] = {0.0, 1.0};
    }
    for (int i = 0; i < NUM_QUAD1; ++i) {
        quads1[i].corners[0] = {0.0, 0.0};
        quads1[i].corners[1] = {0.5, 0.0};
        quads1[i].corners[2] = {0.5, 0.5};
        quads1[i].corners[3] = {0.0, 0.5};
    }

    // Allocate memory for the output array
    double *d_output;
    size_t outputSize = NUM_QUAD0 * NUM_QUAD1 * sizeof(double);
    err = hipMallocManaged(&d_output, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA error (hipMallocManaged for d_output): " << hipGetErrorString(err) << std::endl;
        hipFree(quads0);
        hipFree(quads1);
        return -1;
    }
    hipMallocManaged(&d_output, outputSize);

    // Calculate the number of blocks and threads
    dim3 blockSize(16, 16); // You can tune these numbers based on your GPU capabilities
    dim3 gridSize((NUM_QUAD0 + blockSize.x - 1) / blockSize.x, (NUM_QUAD1 + blockSize.y - 1) / blockSize.y);

    // Launch the kernel
    computeIoUKernel<<<gridSize, blockSize>>>(quads0, quads1, d_output, NUM_QUAD0, NUM_QUAD1);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error (Kernel launch): " << hipGetErrorString(err) << std::endl;
        hipFree(quads0);
        hipFree(quads1);
        hipFree(d_output);
        return -1;
    }
    // Wait for the kernel to complete
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA error (cudaDeviceSynchronize): " << hipGetErrorString(err) << std::endl;
        hipFree(quads0);
        hipFree(quads1);
        hipFree(d_output);
        return -1;
    }
    for (int i = 0; i < 600; i++){
        cout << d_output[i] << endl;
    }
    // Free the allocated managed memory
    hipFree(quads0);
    hipFree(quads1);
    hipFree(d_output);

    return 0;
}
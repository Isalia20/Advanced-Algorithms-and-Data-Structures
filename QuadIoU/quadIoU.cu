#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

struct Point {
    double x, y;
};

struct Edge {
    struct Point startPoint;
    struct Point endPoint;
};

struct Quadrilateral {
    Point corners[4];
};

const Point quadData[][2][4] = {
    {{{0, 0}, {300, 0}, {300, 300}, {0, 300}}, {{0, 0}, {150, 0}, {150, 150}, {0, 150}}},
    {{{0, 0}, {300, 0}, {300, 300}, {0, 300}}, {{500, 500}, {600, 600}, {700, 750}, {600, 750}}},
    {{{200, 200}, {500, 200}, {500, 500}, {200, 500}}, {{300, 300}, {500, 200}, {500, 500}, {200, 500}}},
    {{ {0, 0}, {300, 0}, {300, 300}, {0, 300} }, { {10, 10}, {20, 10}, {20, 20}, {10, 20} }},
    {{ {0, 0}, {300, 0}, {300, 300}, {0, 300} }, { {10, 10}, {20, 10}, {20, 20}, {10, 20} }},
    {{ {0, 0}, {350, 0}, {350, 350}, {0, 350} }, { {15, 15}, {25, 15}, {25, 25}, {15, 25} }},
    {{ {0, 0}, {400, 0}, {400, 400}, {0, 400} }, { {30, 30}, {60, 30}, {60, 60}, {30, 60} }},
    {{ {0, 0}, {450, 0}, {450, 450}, {0, 450} }, { {40, 40}, {80, 40}, {80, 80}, {40, 80} }},
    {{ {0, 0}, {500, 0}, {500, 500}, {0, 500} }, { {50, 50}, {100, 50}, {100, 100}, {50, 100} }},
    {{ {0, 0}, {550, 0}, {550, 550}, {0, 550} }, { {5, 5}, {15, 5}, {15, 15}, {5, 15} }},
    {{ {0, 0}, {600, 0}, {600, 600}, {0, 600} }, { {20, 20}, {40, 20}, {40, 40}, {20, 40} }},
    {{ {0, 0}, {650, 0}, {650, 650}, {0, 650} }, { {25, 25}, {50, 25}, {50, 50}, {25, 50} }},
    {{ {0, 0}, {700, 0}, {700, 700}, {0, 700} }, { {10, 10}, {30, 10}, {30, 30}, {10, 30} }},
    {{ {0, 0}, {750, 0}, {750, 750}, {0, 750} }, { {35, 35}, {70, 35}, {70, 70}, {35, 70} }},
    {{ {0, 0}, {800, 0}, {800, 800}, {0, 800} }, { {45, 45}, {90, 45}, {90, 90}, {45, 90} }},
    {{ {0, 0}, {850, 0}, {850, 850}, {0, 850} }, { {55, 55}, {110, 55}, {110, 110}, {55, 110} }},
    {{ {0, 0}, {900, 0}, {900, 900}, {0, 900} }, { {60, 60}, {120, 60}, {120, 120}, {60, 120} }},
    {{ {0, 0}, {950, 0}, {950, 950}, {0, 950} }, { {65, 65}, {130, 65}, {130, 130}, {65, 130} }},
    {{ {0, 0}, {1000, 0}, {1000, 1000}, {0, 1000} }, { {70, 70}, {140, 70}, {140, 140}, {70, 140} }},
    {{ {0, 0}, {1050, 0}, {1050, 1050}, {0, 1050} }, { {75, 75}, {150, 75}, {150, 150}, {75, 150} }},
    {{ {0, 0}, {1100, 0}, {1100, 1100}, {0, 1100} }, { {80, 80}, {160, 80}, {160, 160}, {80, 160} }},
    {{ {0, 0}, {1150, 0}, {1150, 1150}, {0, 1150} }, { {85, 85}, {170, 85}, {170, 170}, {85, 170} }},
    {{ {0, 0}, {350, 0}, {350, 350}, {0, 350} }, { {15, 15}, {25, 15}, {25, 25}, {15, 25} }},
    {{ {0, 0}, {400, 0}, {400, 400}, {0, 400} }, { {30, 30}, {60, 30}, {60, 60}, {30, 60} }},
    {{ {0, 0}, {450, 0}, {450, 450}, {0, 450} }, { {40, 40}, {80, 40}, {80, 80}, {40, 80} }},
    {{ {0, 0}, {500, 0}, {500, 500}, {0, 500} }, { {50, 50}, {100, 50}, {100, 100}, {50, 100} }},
    {{ {0, 0}, {550, 0}, {550, 550}, {0, 550} }, { {5, 5}, {15, 5}, {15, 15}, {5, 15} }},
    {{ {0, 0}, {600, 0}, {600, 600}, {0, 600} }, { {20, 20}, {40, 20}, {40, 40}, {20, 40} }},
    {{ {0, 0}, {650, 0}, {650, 650}, {0, 650} }, { {25, 25}, {50, 25}, {50, 50}, {25, 50} }},
    {{ {0, 0}, {700, 0}, {700, 700}, {0, 700} }, { {10, 10}, {30, 10}, {30, 30}, {10, 30} }},
    {{ {0, 0}, {750, 0}, {750, 750}, {0, 750} }, { {35, 35}, {70, 35}, {70, 70}, {35, 70} }},
    {{ {0, 0}, {800, 0}, {800, 800}, {0, 800} }, { {45, 45}, {90, 45}, {90, 90}, {45, 90} }},
    {{ {0, 0}, {850, 0}, {850, 850}, {0, 850} }, { {55, 55}, {110, 55}, {110, 110}, {55, 110} }},
    {{ {0, 0}, {900, 0}, {900, 900}, {0, 900} }, { {60, 60}, {120, 60}, {120, 120}, {60, 120} }},
    {{ {0, 0}, {950, 0}, {950, 950}, {0, 950} }, { {65, 65}, {130, 65}, {130, 130}, {65, 130} }},
    {{ {0, 0}, {1000, 0}, {1000, 1000}, {0, 1000} }, { {70, 70}, {140, 70}, {140, 140}, {70, 140} }},
    {{ {0, 0}, {1050, 0}, {1050, 1050}, {0, 1050} }, { {75, 75}, {150, 75}, {150, 150}, {75, 150} }},
    {{ {0, 0}, {1100, 0}, {1100, 1100}, {0, 1100} }, { {80, 80}, {160, 80}, {160, 160}, {80, 160} }},
    {{ {240, 130}, {20, 200}, {200, 20}, {150, 400} }, { {130, 40}, {220, 300}, {220, 375}, {130, 215}}},
    {{ {240, 130}, {20, 200}, {200, 20}, {150, 400} }, { {130, 40}, {220, 300}, {220, 375}, {130, 215}}},
    {{ {200, 20}, {20, 200}, {150, 400}, {240, 130} }, { {130, 40}, {220, 300}, {220, 375}, {130, 215}}}
};

const int NUM_QUADS = sizeof(quadData) / sizeof(quadData[0]);
const int NUM_QUAD0 = NUM_QUADS;
const int NUM_QUAD1 = NUM_QUADS;

void initializeQuads(Quadrilateral* quads, const Point data[][2][4], int numQuads, int quadIndex) {
    for (int i = 0; i < numQuads; ++i) {
        for (int j = 0; j < 4; ++j) {
            quads[i].corners[j] = data[i][quadIndex][j];
        }
    }
}

__device__ double findMaxQuadCoordinate(Quadrilateral& box, char x_or_y){
    // Find the maximum x-coordinate of the quadrilateral
    if (x_or_y == 'x'){
        double max_x = box.corners[0].x; // TODO am here
        for (int i = 1; i < 4; ++i) {
            if (box.corners[i].x > max_x) {
                max_x = box.corners[i].x;
            }
        }
        return max_x;
    } else if (x_or_y == 'y'){
        double max_y = box.corners[0].y;
        for (int i = 1; i < 4; ++i) {
            if (box.corners[i].y > max_y) {
                max_y = box.corners[i].y;
            }
        }
        return max_y;
    } else {
        return -1;
    }
}

__device__ int orientation(Point p, Point q, Point r) {
    double val = (q.y - p.y) * (r.x - q.x) - (q.x - p.x) * (r.y - q.y);

    if (abs(val) < 1e-10) return 0;  // colinear
    return (val > 0) ? 1 : 2;  // clock or counterclock wise
}

__device__ int isPointInsideQuadrilateral(Point point_to_check, Quadrilateral box) {
    // Find the maximum x-coordinate of the quadrilateral
    double max_x = findMaxQuadCoordinate(box, 'x');
    double max_y = findMaxQuadCoordinate(box, 'y');
    // If the point's x-coordinate is greater than the max x-coordinate, it's outside
    if (point_to_check.x > max_x) return -1;
    if (point_to_check.y > max_y) return -1;

    // For each edge of the quadrilateral
    for (int i = 0; i < 4; i++) {
        // Get the current edge's start and end points
        Point start_point = box.corners[i];
        Point end_point = box.corners[(i + 1) % 4]; // Wrap around to the first point after the last
        // Calculate the cross product to determine where the point is in relation to the edge
        double cross_product = (start_point.y - point_to_check.y) * (end_point.x - point_to_check.x) -
                                (start_point.x - point_to_check.x) * (end_point.y - point_to_check.y);
        if (cross_product > 0) {
            return -1; // Point is outside the quadrilateral
        }
        else if (cross_product == 0) {
            return 0; // Point is on the boundary of the quadrilateral
        }
    }
    return 1; // Point is inside the quadrilateral
}

__device__ bool onSegment(Point p, Point q, Point r) {
    return q.x <= max(p.x, r.x) && q.x >= min(p.x, r.x) &&
           q.y <= max(p.y, r.y) && q.y >= min(p.y, r.y);
}

// Helper function to check if two given points are equal
__device__ bool arePointsEqual(const Point& p1, const Point& p2) {
    return abs(p1.x - p2.x) < 1e-10 && abs(p1.y - p2.y) < 1e-10;
}

__device__ bool doIntersect(Point p1, Point q1, Point p2, Point q2, Point& intersection) {
    // Find the four orientations needed for general and
    // special cases
    int o1 = orientation(p1, q1, p2);
    int o2 = orientation(p1, q1, q2);
    int o3 = orientation(p2, q2, p1);
    int o4 = orientation(p2, q2, q1);

    // General case
    if (o1 != o2 && o3 != o4) {
        // Line AB represented as a1x + b1y = c1
        double a1 = q1.y - p1.y;
        double b1 = p1.x - q1.x;
        double c1 = a1 * (p1.x) + b1 * (p1.y);

        // Line CD represented as a2x + b2y = c2
        double a2 = q2.y - p2.y;
        double b2 = p2.x - q2.x;
        double c2 = a2 * (p2.x) + b2 * (p2.y);

        double determinant = a1 * b2 - a2 * b1;

        if (abs(determinant) < 1e-10) {
            return false; // The lines are parallel. This is simplified
                          // by returning false here.
        } else {
            intersection.x = (b2 * c1 - b1 * c2) / determinant;
            intersection.y = (a1 * c2 - a2 * c1) / determinant;
            return true;
        }
    }

    // Special Cases
    // p1, q1 and p2 are colinear and p2 lies on segment p1q1
    if (o1 == 0 && onSegment(p1, p2, q1)) {
        intersection = p2;
        return true;
    }

    // p1, q1 and q2 are colinear and q2 lies on segment p1q1
    if (o2 == 0 && onSegment(p1, q2, q1)) {
        intersection = q2;
        return true;
    }

    // p2, q2 and p1 are colinear and p1 lies on segment p2q2
    if (o3 == 0 && onSegment(p2, p1, q2)) {
        intersection = p1;
        return true;
    }

    // p2, q2 and q1 are colinear and q1 lies on segment p2q2
    if (o4 == 0 && onSegment(p2, q1, q2)) {
        intersection = q1;
        return true;
    }

    return false; // Doesn't fall in any of the above cases
}

__device__ int findIntersectionPoints(Quadrilateral quad_1, Quadrilateral quad_2, Point* intersections, int maxIntersections) {
    int numIntersections = 0;
    for (int i = 0; i < 4; ++i) {
        Edge e1 = {quad_1.corners[i], quad_1.corners[(i + 1) % 4]};
        for (int j = 0; j < 4; ++j) {
            Edge e2 = {quad_2.corners[j], quad_2.corners[(j + 1) % 4]};
            Point intersection;
            if (doIntersect(e1.startPoint, e1.endPoint, e2.startPoint, e2.endPoint, intersection)) {
                // Check if this intersection is already in the intersections array
                bool alreadyExists = false;
                for (int k = 0; k < numIntersections; ++k) {
                    if (arePointsEqual(intersections[k], intersection)) {
                        alreadyExists = true;
                        break;
                    }
                }
                if (!alreadyExists && numIntersections < maxIntersections) {
                    intersections[numIntersections++] = intersection;
                }
            }
        }
    }
    return numIntersections;
}

__device__ int findPointInside(Quadrilateral quad_1, Quadrilateral quad_2, Point* inside_points, int maxPoints) {
    int numInsidePoints = 0;

    for (int i = 0; i < 4; i++) {
        if (isPointInsideQuadrilateral(quad_1.corners[i], quad_2) == 1) {
            if (numInsidePoints < maxPoints) {
                inside_points[numInsidePoints++] = quad_1.corners[i];
            }
        }
        if (isPointInsideQuadrilateral(quad_2.corners[i], quad_1) == 1) {
            if (numInsidePoints < maxPoints) {
                inside_points[numInsidePoints++] = quad_2.corners[i];
            }
        }
    }

    return numInsidePoints;
}

__device__ Point findCentroid(const Point* points, int numPoints) {
    Point centroid = {0.0, 0.0};
    if (numPoints <= 0) {
        return centroid; // Return the origin if there are no points
    }

    for (int i = 0; i < numPoints; ++i) {
        centroid.x += points[i].x;
        centroid.y += points[i].y;
    }
    centroid.x /= (float)numPoints;
    centroid.y /= (float)numPoints;
    return centroid;
}

__device__ double computeAngle(const Point& centroid, const Point& p) {
    return atan2(p.y - centroid.y, p.x - centroid.x);
}

__device__ bool comparePoints(const Point& p1, const Point& p2, const Point& centroid) {
    double angle1 = computeAngle(centroid, p1);
    double angle2 = computeAngle(centroid, p2);

    // Define a small tolerance for angle comparison
    const double TOLERANCE = 1e-6;

    // Check if angles are within the tolerance
    if (fabs(angle1 - angle2) < TOLERANCE) {
        double dist1 = (p1.x - centroid.x) * (p1.x - centroid.x) +
                       (p1.y - centroid.y) * (p1.y - centroid.y);
        double dist2 = (p2.x - centroid.x) * (p2.x - centroid.x) +
                       (p2.y - centroid.y) * (p2.y - centroid.y);
        return dist1 < dist2;
    }
    return angle1 < angle2;
}

__device__ void sortPointsClockwise(Point* points, int numPoints) {
    // First, find the centroid
    Point centroid = findCentroid(points, numPoints);

    // Perform selection sort on the array of points
    for (int i = 0; i < numPoints - 1; i++) {
        // Find the minimum element in unsorted array
        int minIdx = i;
        for (int j = i + 1; j < numPoints; j++) {
            if (comparePoints(points[j], points[minIdx], centroid)) {
                minIdx = j;
            }
        }
        // Swap the found minimum element with the first element
        Point temp = points[minIdx];
        points[minIdx] = points[i];
        points[i] = temp;
    }
}

__device__ void sortPointsClockwise(Quadrilateral& quad) {
    // Directly use the corners array of the quadrilateral for sorting
    sortPointsClockwise(quad.corners, 4);
}

__device__ double polygonArea(const Point* polygon, int n) {
    double area = 0.0;
    // Calculate the sum for the Shoelace formula
    for (int i = 0; i < n - 1; ++i) {
        area += (polygon[i].x * polygon[i + 1].y - polygon[i + 1].x * polygon[i].y);
    }
    // Add the last term (closing the polygon loop)
    area += (polygon[n - 1].x * polygon[0].y - polygon[0].x * polygon[n - 1].y);
    return fabs(area) / 2.0;
}

__device__ double polygonArea(const Quadrilateral& quad) {
    // Since a quadrilateral always has 4 points, we pass the corners directly
    return polygonArea(quad.corners, 4);
}

__device__ double intersectionArea(Quadrilateral& quad0, Quadrilateral& quad1) {
    const int MAX_INTERSECTIONS = 8; // 8 intersections max
    const int MAX_TOTAL_POINTS = MAX_INTERSECTIONS * 2; // Intersection + Inside points
    Point intersectionPoints[MAX_INTERSECTIONS];
    Point insidePoints[MAX_INTERSECTIONS];
    Point allPoints[MAX_TOTAL_POINTS];
    int numIntersectionPoints = 0;
    int numInsidePoints = 0;
    int totalPoints = 0;

    // Sort the corners of both quadrilaterals
    sortPointsClockwise(quad0);
    sortPointsClockwise(quad1);

    // Find intersection points
    numIntersectionPoints = findIntersectionPoints(quad0, quad1, intersectionPoints, MAX_INTERSECTIONS);

    // Find points inside the other quadrilateral
    numInsidePoints = findPointInside(quad0, quad1, insidePoints, MAX_INTERSECTIONS);

    // Combine intersection points with inside points
    for (int i = 0; i < numIntersectionPoints; ++i) {
        allPoints[totalPoints++] = intersectionPoints[i];
    }
    for (int i = 0; i < numInsidePoints; ++i) {
        allPoints[totalPoints++] = insidePoints[i];
    }

    // Sort all points to prepare for area calculation
    sortPointsClockwise(allPoints, totalPoints);

    // Calculate the area of the polygon formed by the points
    double intersectArea;
    if (totalPoints == 0){
        intersectArea = 0;
    } else {
        intersectArea = polygonArea(allPoints, totalPoints);
    }
    return intersectArea;
}

__device__ double unionArea(Quadrilateral& quad0, Quadrilateral& quad1, double intersect_area){
    return polygonArea(quad0) + polygonArea(quad1) - intersect_area;
}

__device__ double calculateIoU(double intersect_area, double union_area){
    return intersect_area / union_area;
}

__global__ void computeIoUKernel(Quadrilateral *quads1, Quadrilateral *quads2, double *output, int n1, int n2) {
    int idx1 = blockIdx.x * blockDim.x + threadIdx.x;
    int idx2 = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx1 < n1 && idx2 < n2) {
        int outputIndex = idx1 * n2 + idx2;
        double areaIntersection = intersectionArea(quads1[idx1], quads2[idx2]);
        double areaUnion = unionArea(quads1[idx1], quads2[idx2], areaIntersection);
        output[outputIndex] = calculateIoU(areaIntersection, areaUnion);
    }
}

__global__ void computeIoUKernelOneToOne(Quadrilateral *quads1, Quadrilateral *quads2, double *output, int numQuads) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < numQuads) {
        double areaIntersection = intersectionArea(quads1[index], quads2[index]);
        double areaUnion = unionArea(quads1[index], quads2[index], areaIntersection);
        output[index] = calculateIoU(areaIntersection, areaUnion);
    }
}

int main() {
    hipError_t err;
    // Allocate memory on the host using Unified Memory (accessible by both CPU and GPU)
    Quadrilateral *quads0, *quads1;
    err = hipMallocManaged(&quads0, NUM_QUAD0 * sizeof(Quadrilateral));
    if (err != hipSuccess) {
        std::cerr << "CUDA error (hipMallocManaged for quads0): " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    err = hipMallocManaged(&quads1, NUM_QUAD1 * sizeof(Quadrilateral));
    if (err != hipSuccess) {
        std::cerr << "CUDA error (hipMallocManaged for quads1): " << hipGetErrorString(err) << std::endl;
        hipFree(quads0); // Clean up previously allocated memory
        return -1;
    }

    // Initialize the arrays with the provided data
    initializeQuads(quads0, quadData, NUM_QUAD0, 0); // Initialize quads0 with the first elements
    initializeQuads(quads1, quadData, NUM_QUAD1, 1); // Initialize quads1 with the second elements

    // Allocate memory for the output array
    double *d_output;
    cout << NUM_QUADS << endl;
    size_t outputSize = NUM_QUADS * sizeof(double);
    err = hipMallocManaged(&d_output, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA error (hipMallocManaged for d_output): " << hipGetErrorString(err) << std::endl;
        hipFree(quads0);
        hipFree(quads1);
        return -1;
    }
    // --------------------------------------------------------------------------------------------------
    // REMOVE THIS BLOCK
    int numQuads = NUM_QUADS; // This should be the number of quadrilaterals in each set
    int threadsPerBlock = 256; // This is an arbitrary number, can be tuned for your GPU
    int blocks = (numQuads + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel with a one-dimensional grid
    computeIoUKernelOneToOne<<<blocks, threadsPerBlock>>>(quads0, quads1, d_output, numQuads);
    // --------------------------------------------------------------------------------------------------
    // // Calculate the number of blocks and threads
    // dim3 blockSize(16, 16); // You can tune these numbers based on your GPU capabilities
    // dim3 gridSize((NUM_QUAD0 + blockSize.x - 1) / blockSize.x, (NUM_QUAD1 + blockSize.y - 1) / blockSize.y);
    // // Launch the kernel
    // computeIoUKernel<<<gridSize, blockSize>>>(quads0, quads1, d_output, NUM_QUAD0, NUM_QUAD1);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error (Kernel launch): " << hipGetErrorString(err) << std::endl;
        hipFree(quads0);
        hipFree(quads1);
        hipFree(d_output);
        return -1;
    }
    // Wait for the kernel to complete
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA error (cudaDeviceSynchronize): " << hipGetErrorString(err) << std::endl;
        hipFree(quads0);
        hipFree(quads1);
        hipFree(d_output);
        return -1;
    }
    for (int i = 0; i < NUM_QUADS; i++){
        cout << d_output[i] << endl;
    }
    // Free the allocated managed memory
    hipFree(quads0);
    hipFree(quads1);
    hipFree(d_output);

    return 0;
}